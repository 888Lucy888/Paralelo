
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>

#define N 9
#define GPUErrorAssertion(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Device function to check if a number is valid in a particular cell
__device__ bool isValid(char* board, int row, int col, char num) {
    // Check row
    for (int i = 0; i < N; ++i) {
        if (board[row * N + i] == num) return false;
    }

    // Check column
    for (int i = 0; i < N; ++i) {
        if (board[i * N + col] == num) return false;
    }

    // Check subgrid
    int startRow = row - row % 3;
    int startCol = col - col % 3;
    for (int i = startRow; i < startRow + 3; ++i) {
        for (int j = startCol; j < startCol + 3; ++j) {
            if (board[i * N + j] == num) return false;
        }
    }

    return true;
}

// Device function to solve the Sudoku puzzle
__global__ void solveSudoku(char* board, bool* solutionFound) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= N * N || *solutionFound) return; // Out of bounds check or solution already found

    while (!(*solutionFound)) {
        if (idx >= N * N) return; // Out of bounds check

        int row = idx / N;
        int col = idx % N;

        if (board[idx] == '.') {
            bool localSolutionFound = false;
            for (char num = '1'; num <= '9'; ++num) {
                if (isValid(board, row, col, num)) {
                    board[idx] = num;
                    localSolutionFound = true;
                    break;
                }
            }
            if (!localSolutionFound) {
                board[idx] = '.';
                return;
            }
        }

        if (idx == N * N - 1) {
            *solutionFound = true;
            return; // Sudoku solved
        }

        idx++;
    }
}

// Host function to print the Sudoku board
// Function: printSudoku
// Description: Prints the solved Sudoku board to the console.
// Parameters:
// - board: The solved Sudoku board represented as a 2D vector of characters.
void printSudoku(const std::vector<char>& board) {
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << board[i * N + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    // Sudoku board
    std::vector<char> board = {
        '5','3','.','.','7','.','.','.','.',
        '6','.','.','1','9','5','.','.','.',
        '.','9','8','.','.','.','.','6','.',
        '8','.','.','.','6','.','.','.','3',
        '4','.','.','8','.','3','.','.','1',
        '7','.','.','.','2','.','.','.','6',
        '.','6','.','.','.','.','2','8','.',
        '.','.','.','4','1','9','.','.','5',
        '.','.','.','.','8','.','.','7','9'
    };

    char* dev_board;
    bool* dev_solutionFound;

    GPUErrorAssertion(hipMalloc((void**)&dev_board, N * N * sizeof(char)));
    GPUErrorAssertion(hipMalloc((void**)&dev_solutionFound, sizeof(bool)));

    GPUErrorAssertion(hipMemcpy(dev_board, board.data(), N * N * sizeof(char), hipMemcpyHostToDevice));

    bool solutionFound = false;

    auto start = std::chrono::steady_clock::now(); // Start time measurement

    solveSudoku<<<1, N * N>>>(dev_board, dev_solutionFound);
    GPUErrorAssertion(hipGetLastError()); // Check for kernel launch errors

    GPUErrorAssertion(hipMemcpy(&solutionFound, dev_solutionFound, sizeof(bool), hipMemcpyDeviceToHost));

    auto end = std::chrono::steady_clock::now(); // End time measurement
    std::chrono::duration<double> elapsed = end - start;

    if (solutionFound) {
        std::vector<char> solvedBoard(N * N);
        GPUErrorAssertion(hipMemcpy(solvedBoard.data(), dev_board, N * N * sizeof(char), hipMemcpyDeviceToHost));
        printSudoku(solvedBoard);
        std::cout << "Elapsed time for GPU solving: " << elapsed.count() << " seconds" << std::endl;
    } else {
        std::cout << "No solution exists for the given Sudoku board.\n";
    }

    GPUErrorAssertion(hipFree(dev_board));
    GPUErrorAssertion(hipFree(dev_solutionFound));

    return 0;
}
